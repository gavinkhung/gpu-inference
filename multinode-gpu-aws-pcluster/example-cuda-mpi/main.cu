#include "hip/hip_runtime.h"
// main.cu
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <mpi.h>

// CUDA kernel to print GPU info
__global__ void printGPUInfo(int rank) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Hello from GPU thread in rank %d (GPU Device: %d)\n", 
               rank, cudaGetDeviceOrdinal());
    }
}

int main(int argc, char** argv) {
    int rank, size, broadcast_value;
    
    // Initialize MPI
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);
    
    // Set GPU device based on local rank
    hipSetDevice(rank % 4);  // Assuming max 4 GPUs per node
    
    // Launch kernel
    printGPUInfo<<<1, 1>>>(rank);
    hipDeviceSynchronize();
    
    // Broadcast example
    if (rank == 0) {
        broadcast_value = 42;
    }
    
    MPI_Bcast(&broadcast_value, 1, MPI_INT, 0, MPI_COMM_WORLD);
    
    printf("Rank %d received broadcast value: %d\n", rank, broadcast_value);
    
    // Cleanup
    hipDeviceSynchronize();
    MPI_Finalize();
    return 0;
}