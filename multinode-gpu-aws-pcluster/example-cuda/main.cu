
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_thread_indices() {
    int threadId = threadIdx.x;
    int blockId = blockIdx.x;
    int globalId = blockId * blockDim.x + threadId;
    
    printf("Thread ID: %d, Block ID: %d, Global ID: %d\n", threadId, blockId, globalId);
}

int main() {
    print_thread_indices<<<2, 4>>>();
    
    hipDeviceSynchronize();
    
    return 0;
}